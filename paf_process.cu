#include "hip/hip_runtime.h"
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <inttypes.h>

#include "multilog.h"
#include "paf_process.cuh"
#include "process.cuh"
#include "cudautil.cuh"

multilog_t *runtime_log;

int main(int argc, char *argv[])
{
  int arg;
  conf_t conf;
  FILE *fp_log = NULL;
  
  /* Setup log interface */
  fp_log = fopen("paf_process.log", "ab+");
  if(fp_log == NULL)
    {
      fprintf(stderr, "Can not open log file paf_process.log\n");
      return EXIT_FAILURE;
    }
  runtime_log = multilog_open("paf_process", 1);
  multilog_add(runtime_log, fp_log);
  multilog(runtime_log, LOG_INFO, "START PAF_PROCESS\n");
  
  /* Initial part */  
  while((arg=getopt(argc,argv,"c:o:i:d:s:h:n:p:r:g:f:b:")) != -1)
    {
      switch(arg)
	{	  
	case 'h':	  	  
	  sscanf(optarg, "%s", conf.hfname);
	  break;

	case 'c':
	  sscanf(optarg, "%lf", &conf.rbufin_ndfstp);
	  break;
	  
	case 's':
	  sscanf(optarg, "%d", &conf.sod);
	  break;
	  
	case 'o':	  
	  if (sscanf (optarg, "%x", &conf.key_out) != 1)
	    {
	      multilog (runtime_log, LOG_ERR, "Could not parse key from %s, which happens at \"%s\", line [%d].\n", optarg, __FILE__, __LINE__);
	      fprintf (stderr, "Could not parse key from %s, which happens at \"%s\", line [%d].\n", optarg, __FILE__, __LINE__);
	      return EXIT_FAILURE;
	    }
	  break;
	  
	case 'i':	  
	  if (sscanf (optarg, "%x", &conf.key_in) != 1)
	    {
	      multilog (runtime_log, LOG_ERR, "Could not parse key from %s, which happens at \"%s\", line [%d].\n", optarg, __FILE__, __LINE__);
	      fprintf (stderr, "Could not parse key from %s, which happens at \"%s\", line [%d].\n", optarg, __FILE__, __LINE__);
	      return EXIT_FAILURE;
	    }
	  break;
	  
	case 'd':
	  sscanf(optarg, "%d", &conf.device_id);
	  break;

	case 'n':
	  sscanf(optarg, "%d", &conf.nstream);
	  break;
	  
	case 'p':
	  sscanf(optarg, "%d", &conf.stream_ndfstp);
	  break;
	  
	case 'g':
	  sscanf(optarg, "%d", &conf.debug);
	  break;
	  
	case 'f':
	  sscanf(optarg, "%s", conf.dir);
	  break;
	  
	case 'b':
	  sscanf(optarg, "%d", &conf.nrun_blk);
	  break;	  
	}
    }
  
#ifdef DEBUG
  struct timespec start, stop;
  double elapsed_time;
  clock_gettime(CLOCK_REALTIME, &start);
#endif
  init_process(&conf);
#ifdef DEBUG
      clock_gettime(CLOCK_REALTIME, &stop);
      elapsed_time = (stop.tv_sec - start.tv_sec) + (stop.tv_nsec - start.tv_nsec)/1000000000.0L;
      fprintf(stdout, "elapsed time for processing prepare is %f s\n\n\n\n\n", elapsed_time);
#endif
      
  /* Check on-board gpus */
//#ifdef DEBUG
//  int deviceCount, device;
//  struct hipDeviceProp_t properties;
//  CudaSafeCall(hipGetDeviceCount(&deviceCount));
//  fprintf(stdout, "Number of devices %d\n", deviceCount);
//  for(device = 0; device < deviceCount; ++device)
//    {
//      hipGetDeviceProperties(&properties, device);
//      if (properties.major != 9999) /* 9999 means emulation only */
//  	{
//  	  printf("multiProcessorCount %d\n",properties.multiProcessorCount);
//  	  printf("maxThreadsPerMultiProcessor %d\n",properties.maxThreadsPerMultiProcessor);
//  	  printf("pciDeviceID %d\n",properties.pciDeviceID);
//  	  printf("pciBusID %d\n",properties.pciBusID);
//  	}
//    }
//#endif
  
  /* Play with data */
#ifdef DEBUG
  clock_gettime(CLOCK_REALTIME, &start);
#endif
  if(do_process(conf))
    {
      multilog (runtime_log, LOG_ERR, "Can not finish the process, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "Can not finish the process, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  #ifdef DEBUG
      clock_gettime(CLOCK_REALTIME, &stop);
      elapsed_time = (stop.tv_sec - start.tv_sec) + (stop.tv_nsec - start.tv_nsec)/1000000000.0L;
      fprintf(stdout, "elapsed time for data processing is %f s\n", elapsed_time);
#endif

  destroy_process(conf);

  /* Destory log interface */
  multilog(runtime_log, LOG_INFO, "FINISH PAF_PROCESS\n\n");
  multilog_close(runtime_log);
  fclose(fp_log);
  
  return EXIT_SUCCESS;
}